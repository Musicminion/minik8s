#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;
const int M = 8;
const int N = 8;

__global__ void matrix_add(int **A, int **B, int **C) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    int j = (blockIdx.y * blockDim.y + threadIdx.y);
    C[i][j] = A[i][j] + B[i][j];
}

int main() {
    int nbytes=M*N*sizeof(int);
    //这两个是位于host机上的
    int **host_A = (int **) malloc(M * sizeof(int *));
    int **host_B = (int **) malloc(M * sizeof(int *));
    int **host_C = (int **) malloc(M * sizeof(int *));
    int *data_A = (int *) malloc(nbytes);
    int *data_B = (int *) malloc(nbytes);
    int *data_C = (int *) malloc(nbytes);
    for (int i = 0; i < M; i++) {
        host_A[i] = &data_A[i * N];
        host_B[i] = &data_B[i * N];
        host_C[i] = &data_C[i * N];
        for (int j = 0; j < N ; j++) {
            data_A[i*N+j] = i*N+j;
            data_B[i*N+j] = i*N+j;
            data_C[i*N+j] = 0;
        }
    }

    //这里说明了在host上的指针组成的数组是好的
    cout<<"host上面的矩阵A:"<<endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            cout<<host_A[i][j]<<" ";
        }
        cout<<endl;
    }

    cout<<"host上面的矩阵B:"<<endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            cout<<host_B[i][j]<<" ";
        }
        cout<<endl;
    }

    int **dev_A, **dev_B, **dev_C;
    int *dev_A1, *dev_B1, *dev_C1;
    hipMalloc((void **)&dev_A1, nbytes);
    hipMalloc((void **)&dev_B1, nbytes);
    hipMalloc((void **)&dev_C1, nbytes);
    //数据拷贝
    hipMemcpy((void *)dev_A1, (void *)data_A, nbytes, hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_B1, (void *)data_B, nbytes, hipMemcpyHostToDevice);

    hipMemset((void *)dev_C1, 0, nbytes);
    for (int i = 0; i < M; i++) {
        host_A[i] = dev_A1 + i * N;
        host_B[i] = dev_B1 + i * N;
        host_C[i] = dev_C1 + i * N;
    }

    hipMalloc((void **)&dev_A, sizeof(int *) * M);
    hipMalloc((void **)&dev_B, sizeof(int *) * M);
    hipMalloc((void **)&dev_C, sizeof(int *) * M);

    hipMemcpy((void *)dev_A, (void *)host_A, sizeof(int *) * M, hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_B, (void *)host_B, sizeof(int *) * M, hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_C, (void *)host_C, sizeof(int *) * M, hipMemcpyHostToDevice);


    dim3 grid(M / 2, N / 2);
    dim3 block(2, 2);
    matrix_add<<<grid, block>>>(dev_A, dev_B, dev_C);

    hipMemcpy((void *) data_C,(void *) dev_C1, nbytes, hipMemcpyDeviceToHost);

    cout<<"矩阵加法的结果:"<<endl;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            cout<<data_C[i*N+j]<<" ";
        }
        cout<<endl;
    }
    free(data_A);
    free(data_B);
    free(data_C);
    free(host_A);
    free(host_B);
    free(host_C);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipFree(dev_A1);
    hipFree(dev_B1);
    hipFree(dev_C1);

    return 0;
}