#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#define get_tid_x() (blockIdx.x * blockDim.x + threadIdx.x)
#define get_tid_y() (blockIdx.y * blockDim.y + threadIdx.y)

const int M = 8;
const int N = 8;

static void HandleError(hipError_t err,const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        cout<<endl;
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

//核函数
__global__ void matrix_add(int **A, int **B, int **C) {
    int i = get_tid_x();
    int j = get_tid_y();  
    C[i][j] = A[i][j] + B[i][j];
}

int main() {
    int nbytes=M*N*sizeof(int);
    
    int **A = (int **) malloc(sizeof(int *) * M);
    int **B = (int **) malloc(sizeof(int *) * M);
    int **C = (int **) malloc(sizeof(int *) * M);

    int *data_A = (int *) malloc(sizeof(int) * M * N);
    int *data_B = (int *) malloc(sizeof(int) * M * N);
    int *data_C = (int *) malloc(sizeof(int) * M * N);


    //这里使用了强制类型转换
    int *host_A = (int *) malloc(nbytes);
    int *host_B = (int *) malloc(nbytes);
    int *host_C = (int *) malloc(nbytes);


    for (int i = 0; i < M * N; i++) {
        data_A[i] = i;
        data_B[i] = i;
        host_A[i] = i;
        host_B[i] = i;
    }

    printf("Matrix A is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_A[i * N + j]);
        }
        printf("\n");
    }

    printf("Matrix B is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_B[i * N + j]);
        }
        printf("\n");
    }

    int *dev_data_A;
    int *dev_data_B;
    int *dev_data_C;

    // malloc matrix (size = M*N) in GPU device
    HANDLE_ERROR(hipMalloc((void **) &dev_data_A, sizeof(int) * M * N));
    HANDLE_ERROR(hipMalloc((void **) &dev_data_B, sizeof(int) * M * N));
    HANDLE_ERROR(hipMalloc((void **) &dev_data_C, sizeof(int) * M * N));

    // copy data from host to GPU device
    HANDLE_ERROR(hipMemcpy((void *) dev_data_A, (void *) data_A, sizeof(int) * M * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_data_B, (void *) data_B, sizeof(int) * M * N, hipMemcpyHostToDevice));
    // init C
    HANDLE_ERROR(hipMemset((void *) dev_data_C, 0, sizeof(int) * M * N));

    for (int i = 0; i < M; i++) {
        A[i] = dev_data_A + i * N;
        B[i] = dev_data_B + i * N;
        C[i] = dev_data_C + i * N;
    }

    int **dev_A;
    int **dev_B;
    int **dev_C;

    HANDLE_ERROR(hipMalloc((void **) &dev_A, sizeof(int *) * M));
    HANDLE_ERROR(hipMalloc((void **) &dev_B, sizeof(int *) * M));
    HANDLE_ERROR(hipMalloc((void **) &dev_C, sizeof(int *) * M));

    HANDLE_ERROR(hipMemcpy((void *) dev_A, (void *) A, sizeof(int *) * M, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_B, (void *) B, sizeof(int *) * M, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_C, (void *) C, sizeof(int *) * M, hipMemcpyHostToDevice));

    dim3 threadPerBlock(5, 5);
    dim3 numBlocks(M / threadPerBlock.x, N / threadPerBlock.y);

    matrix_add <<<numBlocks, threadPerBlock>>> (dev_A, dev_B, dev_C);

    // copy result to host
    HANDLE_ERROR(hipMemcpy((void *) data_C, (void *) dev_data_C, sizeof(int) * M * N, hipMemcpyDeviceToHost));

    // print result: 
    printf("The matrix add result is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_C[i * N + j]);
        }
        printf("\n");
    }
}