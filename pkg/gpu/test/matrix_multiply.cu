#include <stdio.h>
#include "hip/hip_runtime.h"

// Matrix size: 50 * 25 & 25 * 50
const int M = 15;
const int N = 10;

static void HandleError(hipError_t err,const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

// Matrix multiply: C = A * B
__global__ void matrix_multiply(int **A, int **B, int **C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;  
    C[i][j] = A[i][j] * B[j][i];
}

int main() {
    int count;
    hipGetDeviceCount(&count);
    printf("gpu num %d\n", count);
    
    int **A = (int **) malloc(sizeof(int *) * M);
    int **B = (int **) malloc(sizeof(int *) * N);
    int **C = (int **) malloc(sizeof(int *) * M);

    int *data_A = (int *) malloc(sizeof(int) * M * N);
    int *data_B = (int *) malloc(sizeof(int) * M * N);
    int *data_C = (int *) malloc(sizeof(int) * M * N);
    for (int i = 0; i < M * N; i++) {
        data_A[i] = i;
        data_B[i] = i;
    }

    printf("Matrix A is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_A[i * N + j]);
        }
        printf("\n");
    }

    printf("Matrix B is:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M ; j++) {
            printf("%d ", data_B[i * M + j]);
        }
        printf("\n");
    }

    int *dev_data_A;
    int *dev_data_B;
    int *dev_data_C;

    // malloc matrix (size = M*N) in GPU device
    HANDLE_ERROR(hipMalloc((void **) &dev_data_A, sizeof(int) * M * N));
    HANDLE_ERROR(hipMalloc((void **) &dev_data_B, sizeof(int) * M * N));
    HANDLE_ERROR(hipMalloc((void **) &dev_data_C, sizeof(int) * M * M));

    // copy data from host to GPU device
    HANDLE_ERROR(hipMemcpy((void *) dev_data_A, (void *) data_A, sizeof(int) * M * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_data_B, (void *) data_B, sizeof(int) * M * N, hipMemcpyHostToDevice));
    // init C
    HANDLE_ERROR(hipMemset((void *) dev_data_C, 0, sizeof(int) * M * M));

    for (int i = 0; i < M; i++) {
        A[i] = dev_data_A + i * N;
        C[i] = dev_data_C + i * M;
    }

    for (int i = 0; i < N; i++) {
        B[i] = dev_data_B + i * M;
    }

    int **dev_A;
    int **dev_B;
    int **dev_C;

    HANDLE_ERROR(hipMalloc((void **) &dev_A, sizeof(int *) * M));
    HANDLE_ERROR(hipMalloc((void **) &dev_B, sizeof(int *) * N));
    HANDLE_ERROR(hipMalloc((void **) &dev_C, sizeof(int *) * M));

    HANDLE_ERROR(hipMemcpy((void *) dev_A, (void *) A, sizeof(int *) * M, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_B, (void *) B, sizeof(int *) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) dev_C, (void *) C, sizeof(int *) * M, hipMemcpyHostToDevice));

    dim3 threadPerBlock(5, 5);
    dim3 numBlocks(M / threadPerBlock.x, N / threadPerBlock.y);

    matrix_multiply <<<numBlocks, threadPerBlock>>> (dev_A, dev_B, dev_C);

    // copy result to host
    HANDLE_ERROR(hipMemcpy((void *) data_C, (void *) dev_data_C, sizeof(int) * M * M, hipMemcpyDeviceToHost));

    // print result: 
    printf("The matrix multiply result is:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N ; j++) {
            printf("%d ", data_C[i * M + j]);
        }
        printf("\n");
    }
}